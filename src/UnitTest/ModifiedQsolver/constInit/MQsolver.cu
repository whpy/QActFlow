#include "hip/hip_runtime.h"
#include "MQsolver.cuh"

int main(){
    // computation parameters
    int BSZ = 16;
    int Ns = 200000;
    int Nx = 768; // same as colin
    int Ny = 768;
    int Nxh = Nx/2+1;
    Qreal Lx = 33 * 2 *M_PI;
    Qreal Ly = Lx;
    Qreal dx = Lx/Nx;
    Qreal dy = dx;
    Qreal dt = 0.0001; // same as colin

    // non-dimensional number
    Qreal Re = 0.1;
    Qreal Er = 0.1;
    Qreal Rf = 7.5 * 0.00001;
    Qreal lambda = 0.1;
    Qreal ra = 0.2;

    // main Fields to be solved
    // *_curr act as an intermediate while RK4 timeintegration
    // *_new store the value of next time step 

    Mesh *mesh = new Mesh(BSZ, Nx, Ny, Lx, Ly);
    cout << "Re = " << Re << endl;
    cout << "Er = " << Er << endl;
    cout << "lambda = " << lambda << endl;
    cout << "Rf = " << Rf << endl; cout << "Ra = " << ra << endl;
    cout<< "Lx: " << mesh->Lx << " "<< "Ly: " << mesh->Ly << " " << endl;
    cout<< "Nx: " << mesh->Nx << " "<< "Ny: " << mesh->Ny << " " << endl;
    cout<< "dx: " << mesh->dx << " "<< "dy: " << mesh->dy << " " << endl;
    cout<< "Nx*dx: " << mesh->Nx*mesh->dx << " "<< "Ny*dy: " << mesh->Ny*mesh->dy << " " << endl;
    Field *w_old = new Field(mesh); Field *w_curr = new Field(mesh); Field *w_new = new Field(mesh);
    Field *r1_old = new Field(mesh); Field *r1_curr = new Field(mesh); Field *r1_new = new Field(mesh);
    Field *r2_old = new Field(mesh); Field *r2_curr = new Field(mesh); Field *r2_new = new Field(mesh);

     // linear factors
    Qreal *wIF, *wIFh; Qreal *r1IF, *r1IFh; Qreal *r2IF, *r2IFh;
    hipMallocManaged(&wIF, sizeof(Qreal)*Nxh*Ny); hipMallocManaged(&wIFh, sizeof(Qreal)*Nxh*Ny);
    hipMallocManaged(&r1IF, sizeof(Qreal)*Nxh*Ny); hipMallocManaged(&r1IFh, sizeof(Qreal)*Nxh*Ny);
    hipMallocManaged(&r2IF, sizeof(Qreal)*Nxh*Ny); hipMallocManaged(&r2IFh, sizeof(Qreal)*Nxh*Ny);

    // intermediate fields
    //nonlinear terms
    Field *wnonl = new Field(mesh); Field *r1nonl = new Field(mesh); Field *r2nonl = new Field(mesh);
    // velocity and S
    Field *u = new Field(mesh); Field *v = new Field(mesh); Field *S = new Field(mesh);
    // H tensor
    Field *h11 = new Field(mesh); Field *h12 = new Field(mesh);
    // the stress tensor
    Field *p11 = new Field(mesh); Field *p12 = new Field(mesh); Field* p21 = new Field(mesh);
    // auxiliary fields
    Field *aux = new Field(mesh); Field *aux1 = new Field(mesh); 

    // field \alpha to be modified (scalar at the very first)
    Field *Ra = new Field(mesh);

    //////////////////////// precomputation //////////////////////////
    r1lin_func<<<mesh->dimGridsp, mesh->dimBlocksp>>>(r1IFh, r1IF, dt, r1_old->mesh->Nxh, r1_old->mesh->Ny, r1_old->mesh->BSZ);
    r2lin_func<<<mesh->dimGridsp, mesh->dimBlocksp>>>(r2IFh, r2IF, dt, r2_old->mesh->Nxh, r2_old->mesh->Ny, r2_old->mesh->BSZ);
    wlin_func<<<mesh->dimGridsp, mesh->dimBlocksp>>>(wIFh, wIF, w_old->mesh->k_squared, Re, Rf, dt, w_old->mesh->Nxh, w_old->mesh->Ny, w_old->mesh->BSZ);

    // the precomputation function also updates the spectrum of corresponding variables
    precompute_func(r1_old, r2_old, w_old, Phy_init);
    Ra_init(Ra->phys, ra, dx, dy, Nx, Ny);
    FwdTrans(mesh, Ra->phys, Ra->spec);
    cuda_error_func( hipDeviceSynchronize() );
    coord(*mesh);
    field_visual(w_old, "wstart.csv");
    field_visual(r1_old, "r1start.csv");
    field_visual(r2_old, "r2start.csv");

    for(int m=0 ;m<Ns ;m++){
        integrate_func0(w_old, w_curr, w_new, wIF, wIFh, dt);
        integrate_func0(r1_old, r1_curr, r1_new, r1IF, r1IFh, dt);
        integrate_func0(r2_old, r2_curr, r2_new, r2IF, r2IFh, dt);
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r1_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r2_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(w_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        
        curr_func(r1_curr, r2_curr, w_curr, u, v, S, h11, h12);
        wnonl_func(wnonl, aux, aux1, p11, p12, p21, 
        r1_curr, r2_curr, w_curr, u, v, Ra, S, Re, Er, lambda, aux, aux1);
        r1nonl_func(r1nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, aux, aux1);
        r2nonl_func(r2nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, aux, aux1);
        integrate_func1(w_old, w_curr, w_new, wnonl, wIF, wIFh, dt);
        integrate_func1(r1_old, r1_curr, r1_new, r1nonl, r1IF, r1IFh, dt);
        integrate_func1(r2_old, r2_curr, r2_new, r2nonl, r2IF, r2IFh, dt);
       
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r1_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r2_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(w_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        curr_func(r1_curr, r2_curr, w_curr, u, v, S, h11, h12);
        wnonl_func(wnonl, aux, aux1, p11, p12, p21, 
        r1_curr, r2_curr, w_curr, u, v, Ra, S, Re, Er, lambda, aux, aux1);
        r1nonl_func(r1nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, aux, aux1);
        r2nonl_func(r2nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, aux, aux1);
        integrate_func2(w_old, w_curr, w_new, wnonl, wIF, wIFh, dt);
        integrate_func2(r1_old, r1_curr, r1_new, r1nonl, r1IF, r1IFh, dt);
        integrate_func2(r2_old, r2_curr, r2_new, r2nonl, r2IF, r2IFh, dt);
        
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r1_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r2_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(w_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        curr_func(r1_curr, r2_curr, w_curr, u, v, S, h11, h12);
        wnonl_func(wnonl, aux, aux1, p11, p12, p21, 
        r1_curr, r2_curr, w_curr, u, v, Ra, S, Re, Er, lambda, aux, aux1);
        r1nonl_func(r1nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, aux, aux1);
        r2nonl_func(r2nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, aux, aux1);
        integrate_func3(w_old, w_curr, w_new, wnonl, wIF, wIFh, dt);
        integrate_func3(r1_old, r1_curr, r1_new, r1nonl, r1IF, r1IFh, dt);
        integrate_func3(r2_old, r2_curr, r2_new, r2nonl, r2IF, r2IFh, dt);
        
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r1_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r2_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        // dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(w_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        curr_func(r1_curr, r2_curr, w_curr, u, v, S, h11, h12);
        wnonl_func(wnonl, aux, aux1, p11, p12, p21, 
        r1_curr, r2_curr, w_curr, u, v, Ra, S, Re, Er, lambda, aux, aux1);
        r1nonl_func(r1nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, aux, aux1);
        r2nonl_func(r2nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, aux, aux1);
        integrate_func4(w_old, w_curr, w_new, wnonl, wIF, wIFh, dt);
        integrate_func4(r1_old, r1_curr, r1_new, r1nonl, r1IF, r1IFh, dt);
        integrate_func4(r2_old, r2_curr, r2_new, r2nonl, r2IF, r2IFh, dt);

        cuda_error_func( hipDeviceSynchronize() );
        SpecSet<<<mesh->dimGridsp, mesh->dimBlocksp>>>(w_old->spec, w_new->spec, 
        w_old->mesh->Nxh, w_old->mesh->Ny, w_old->mesh->BSZ);

        SpecSet<<<mesh->dimGridsp, mesh->dimBlocksp>>>(r1_old->spec, r1_new->spec, 
        r1_old->mesh->Nxh, r1_old->mesh->Ny, r1_old->mesh->BSZ);

        SpecSet<<<mesh->dimGridsp, mesh->dimBlocksp>>>(r2_old->spec, r2_new->spec, 
        r2_old->mesh->Nxh, r2_old->mesh->Ny, r2_old->mesh->BSZ);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r1_old->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r2_old->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(w_old->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);

        if (m%10 == 0) {

        cout << "\r" << "t = " << m*dt << flush;
        }
        if (m%200 == 0){
            BwdTrans(mesh, r1_old->spec, r1_old->phys);
            BwdTrans(mesh, r2_old->spec, r2_old->phys);
            BwdTrans(mesh, w_old->spec, w_old->phys);
            S_func(r1_old, r2_old, S);
            cuda_error_func( hipDeviceSynchronize() );

            field_visual(r1_old, to_string(m)+"r1.csv");
            field_visual(r2_old, to_string(m)+"r2.csv");
            field_visual(w_old, to_string(m)+"w.csv");
            field_visual(S, to_string(m)+"S.csv");
            if (std::isnan(r1_old->phys[0])) {"NAN ";exit(0);}
        }
}
}

