#include "hip/hip_runtime.h"
#include "Qsolver.cuh"
using std::string;
using std::cout; 
using std::endl;



// void r1_init(Qreal *r1, Qreal dx, Qreal dy, int Nx, int Ny){
//     for (int j=0; j<Ny; j++){
//         for (int i=0; i<Nx; i++){
//             int index = i+j*Nx;
//             float x = dx*i;
//             float y = dy*j;
//             r1[index] = (float(rand())/RAND_MAX)-0.5;
//         }
//     }
// }

// void r2_init(Qreal *r2, Qreal dx, Qreal dy, int Nx, int Ny){
//     for (int j=0; j<Ny; j++){
//         for (int i=0; i<Nx; i++){
//             int index = i+j*Nx;
//             float x = dx*i;
//             float y = dy*j;
//             r2[index] = (float(rand())/RAND_MAX)-0.5;
//         }
//     }
// }


// void w_init(Qreal *w, Qreal dx, Qreal dy, int Nx, int Ny){
//     for (int j=0; j<Ny; j++){
//         for (int i=0; i<Nx; i++){
//             int index = i+j*Nx;
//             w[index] = 0.000;
//         }
//     }
// }

// void alpha_init(Qreal *alpha, Qreal Ra, Qreal dx, Qreal dy, int Nx, int Ny){
//     for (int j=0; j<Ny; j++){
//         for (int i=0; i<Nx; i++){
//             int index = i+j*Nx;
//             alpha[index] = (Ra);
//         }
//     }
// }
// void precompute_func(Field* r1, Field* r2, Field* w, InitType flag){
//     Mesh* mesh = r1->mesh;
//     int Nx = mesh->Nx; int Ny = mesh->Ny;
//     Qreal dx = mesh->dx; Qreal dy = mesh->dy;

//     if (flag == Def_init){
//         r1_init(r1->phys, dx, dy, Nx, Ny);
//         r2_init(r2->phys, dx, dy, Nx, Ny);
//         w_init(w->phys, dx, dy, Nx, Ny);
//     }
//     else if (flag == File_init){
//         file_init("./init/r1_init.csv", r1);
//         file_init("./init/r2_init.csv", r2);
//         file_init("./init/w_init.csv", w);
//     }

//     FwdTrans(mesh, r1->phys, r1->spec);
//     FwdTrans(mesh, r2->phys, r2->spec);
//     FwdTrans(mesh, w->phys, w->spec);
// }


int main(){
    int startpoint = 19980;
    // computation parameters
    int BSZ = 16;
    int Ns = 40000;
    int Nx = 512; // same as colin
    int Ny = 512;
    int Nxh = Nx/2+1;
    Qreal Lx = 33 * 2 *M_PI;
    Qreal Ly = Lx;
    Qreal dx = Lx/Nx;
    Qreal dy = dx;
    Qreal dt = 0.001; // same as colin
    // Qreal a = 1.0;

    //////////////////////// variables definitions //////////////////////////

    // non-dimensional number in "The role of advective inertia in active nematic"  
    double Re_n = 0.1;
    // to distinguish the Er we use in theis programe
    // we denotes the Er in paper as Er_n
    double Er_n = 0.1;
    // Ra = (ln/la)^2, which the square of \tiled(alpha), noted that it assumes to be negative
    // and we assume to be positive 
    double Ra = -0.2; 
    // Rf = (ln/lf)^2, which we defines cf = lc/lf where lc = ln in prl.
    // so cf = sqrt(Rf)
    double Rf = 7.5*0.00001;
    Qreal lambda = 0.1;
    
    // C_{cn} = lc/ln = 1.0 as colin set lc = ln.
    // Qreal cn = lc/ln;
    Qreal cn = 1.0;
    // C_{cf} = lc/lf = ln/lf = sqrt(Rf)
    // Qreal cf = lc/lf;
    Qreal cf = sqrt(Rf);
    Qreal Er = Er_n;
    Qreal Re = Re_n;

    // by specially choosing the scales, colin makes the Pe equals to 1.0
    Qreal Pe = 1.0;
    
    // main Fields to be solved
    // *_curr act as an intermediate while RK4 timeintegration
    // *_new store the value of next time step 

    Mesh *mesh = new Mesh(BSZ, Nx, Ny, Lx, Ly);
    cout << "start point: " << startpoint << endl;
    cout << "Re = " << Re << endl;
    cout << "Er = " << Er << endl;
    cout << "Ra = " << Ra << endl;
    cout << "Pe = " << Pe << endl;
    cout << "cf = " << cf << endl;
    cout<< "Lx: " << mesh->Lx << " "<< "Ly: " << mesh->Ly << " " << endl;
    cout<< "Nx: " << mesh->Nx << " "<< "Ny: " << mesh->Ny << " " << endl;
    cout<< "dx: " << mesh->dx << " "<< "dy: " << mesh->dy << " " << endl;
    cout<< "Nx*dx: " << mesh->Nx*mesh->dx << " "<< "Ny*dy: " << mesh->Ny*mesh->dy << " " << endl;
    Field *w_old = new Field(mesh); Field *w_curr = new Field(mesh); Field *w_new = new Field(mesh);
    Field *r1_old = new Field(mesh); Field *r1_curr = new Field(mesh); Field *r1_new = new Field(mesh);
    Field *r2_old = new Field(mesh); Field *r2_curr = new Field(mesh); Field *r2_new = new Field(mesh);
    
    // linear factors
    Qreal *wIF, *wIFh; Qreal *r1IF, *r1IFh; Qreal *r2IF, *r2IFh;
    hipMallocManaged(&wIF, sizeof(Qreal)*Nxh*Ny); hipMallocManaged(&wIFh, sizeof(Qreal)*Nxh*Ny);
    hipMallocManaged(&r1IF, sizeof(Qreal)*Nxh*Ny); hipMallocManaged(&r1IFh, sizeof(Qreal)*Nxh*Ny);
    hipMallocManaged(&r2IF, sizeof(Qreal)*Nxh*Ny); hipMallocManaged(&r2IFh, sizeof(Qreal)*Nxh*Ny);

    // intermediate fields
    Field *wnonl = new Field(mesh); Field *r1nonl = new Field(mesh); Field *r2nonl = new Field(mesh);
    Field *u = new Field(mesh); Field *v = new Field(mesh); Field *S = new Field(mesh);
    Field *p11 = new Field(mesh); Field *p12 = new Field(mesh); Field* p21 = new Field(mesh);
    
    // auxiliary fields
    Field *aux = new Field(mesh); Field *aux1 = new Field(mesh); 

    // field \alpha to be modified (scalar at the very first)
    Field *alpha = new Field(mesh);

    //////////////////////// precomputation //////////////////////////
    r1lin_func<<<mesh->dimGridsp, mesh->dimBlocksp>>>(r1IFh, r1IF, r1_old->mesh->k_squared, Re, cn, dt, r1_old->mesh->Nxh, r1_old->mesh->Ny, r1_old->mesh->BSZ);
    r2lin_func<<<mesh->dimGridsp, mesh->dimBlocksp>>>(r2IFh, r2IF, r2_old->mesh->k_squared, Re, cn, dt, r2_old->mesh->Nxh, r2_old->mesh->Ny, r2_old->mesh->BSZ);
    wlin_func<<<mesh->dimGridsp, mesh->dimBlocksp>>>(wIFh, wIF, w_old->mesh->k_squared, Re, cf, dt, w_old->mesh->Nxh, w_old->mesh->Ny, w_old->mesh->BSZ);
    
    // the precomputation function also updates the spectrum of corresponding variables
    precompute_func(r1_old, r2_old, w_old, File_init);
    alpha_init(alpha->phys, Ra, dx, dy, Nx, Ny);
    FwdTrans(mesh, alpha->phys, alpha->spec);
    // prepare the referenced system
    cuda_error_func( hipDeviceSynchronize() );
    field_visual(w_old,"wstart.csv");
    field_visual(r1_old,"r1start.csv");
    field_visual(r2_old,"r2start.csv");
    coord(*mesh);
    
    //////////////////////// time stepping //////////////////////////
    for(int m=0 ;m<Ns ;m++){
        // cout << "flag 6" << endl;
        curr_func(r1_curr, r2_curr, w_curr, u, v, S);
        integrate_func0(w_old, w_curr, w_new, wIF, wIFh, dt);
        integrate_func0(r1_old, r1_curr, r1_new, r1IF, r1IFh, dt);
        integrate_func0(r2_old, r2_curr, r2_new, r2IF, r2IFh, dt);
        // cout << "flag 7" << endl;
        // cuda_error_func( hipDeviceSynchronize() );
        // BwdTrans(mesh, ucurr->spec, ucurr->phys);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r1_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r2_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(w_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        curr_func(r1_curr, r2_curr, w_curr, u, v, S);
        // cout << "flag 9" << endl;
        wnonl_func(wnonl, aux, aux1, p11, p12, p21, r1_curr, r2_curr, w_curr, u, v, alpha, S, Re, Er, cn, lambda);
        // cout  << "flag 11" << endl;
        r1nonl_func(r1nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, cn, Pe);
        r2nonl_func(r2nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, cn, Pe);
        // cout  << "flag 10" << endl;
        integrate_func1(w_old, w_curr, w_new, wnonl, wIF, wIFh, dt);
        integrate_func1(r1_old, r1_curr, r1_new, r1nonl, r1IF, r1IFh, dt);
        integrate_func1(r2_old, r2_curr, r2_new, r2nonl, r2IF, r2IFh, dt);
        // cout << "flag 8" << endl;
        // cuda_error_func( hipDeviceSynchronize() );
        // unonl_func(unonl, ucurr, m*dt);
        // cuda_error_func( hipDeviceSynchronize() );
        // integrate_func1(u, ucurr, unew, unonl, IFu, IFuh, dt);
        // BwdTrans(mesh, ucurr->spec, ucurr->phys);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r1_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r2_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(w_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        curr_func(r1_curr, r2_curr, w_curr, u, v, S);
        wnonl_func(wnonl, aux, aux1, p11, p12, p21, r1_curr, r2_curr, w_curr, u, v, alpha, S, Re, Er, cn, lambda);
        r1nonl_func(r1nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, cn, Pe);
        r2nonl_func(r2nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, cn, Pe);
        integrate_func2(w_old, w_curr, w_new, wnonl, wIF, wIFh, dt);
        integrate_func2(r1_old, r1_curr, r1_new, r1nonl, r1IF, r1IFh, dt);
        integrate_func2(r2_old, r2_curr, r2_new, r2nonl, r2IF, r2IFh, dt);
        
        // cuda_error_func( hipDeviceSynchronize() );
        // unonl_func(unonl, ucurr, m*dt);
        // integrate_func2(u, ucurr, unew, unonl, IFu, IFuh, dt);
        // BwdTrans(mesh, ucurr->spec, ucurr->phys);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r1_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r2_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(w_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        curr_func(r1_curr, r2_curr, w_curr, u, v, S);
        wnonl_func(wnonl, aux, aux1, p11, p12, p21, r1_curr, r2_curr, w_curr, u, v, alpha, S, Re, Er, cn, lambda);
        r1nonl_func(r1nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, cn, Pe);
        r2nonl_func(r2nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, cn, Pe);
        integrate_func3(w_old, w_curr, w_new, wnonl, wIF, wIFh, dt);
        integrate_func3(r1_old, r1_curr, r1_new, r1nonl, r1IF, r1IFh, dt);
        integrate_func3(r2_old, r2_curr, r2_new, r2nonl, r2IF, r2IFh, dt);
        
        // cuda_error_func( hipDeviceSynchronize() );
        // unonl_func(unonl, ucurr, m*dt);
        // integrate_func3(u, ucurr, unew, unonl, IFu, IFuh, dt);
        // BwdTrans(mesh, ucurr->spec, ucurr->phys);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r1_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(r2_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        dealiasing_func<<<mesh->dimBlocksp, mesh->dimBlocksp>>>(w_curr->spec, mesh->cutoff, mesh->Nxh, mesh->Ny, mesh->BSZ);
        curr_func(r1_curr, r2_curr, w_curr, u, v, S);
        wnonl_func(wnonl, aux, aux1, p11, p12, p21, r1_curr, r2_curr, w_curr, u, v, alpha, S, Re, Er, cn, lambda);
        r1nonl_func(r1nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, cn, Pe);
        r2nonl_func(r2nonl, aux, r1_curr, r2_curr, w_curr, u, v, S, lambda, cn, Pe);
        integrate_func4(w_old, w_curr, w_new, wnonl, wIF, wIFh, dt);
        integrate_func4(r1_old, r1_curr, r1_new, r1nonl, r1IF, r1IFh, dt);
        integrate_func4(r2_old, r2_curr, r2_new, r2nonl, r2IF, r2IFh, dt);
        curr_func(r1_curr, r2_curr, w_curr, u, v, S);
        // cuda_error_func( hipDeviceSynchronize() );
        // unonl_func(unonl, ucurr, m*dt);
        // integrate_func4(u, ucurr, unew, unonl, IFu, IFuh, dt);
        // BwdTrans(mesh, ucurr->spec, ucurr->phys);
        // cuda_error_func( hipDeviceSynchronize() );
        // unonl_func(unonl, ucurr, m*dt);
        // cout << "flag 8" << endl;
        cuda_error_func( hipDeviceSynchronize() );
        SpecSet<<<mesh->dimGridsp, mesh->dimBlocksp>>>(w_old->spec, w_new->spec, w_old->mesh->Nxh, w_old->mesh->Ny, w_old->mesh->BSZ);
        // cout << "flag 13" << endl;
        SpecSet<<<mesh->dimGridsp, mesh->dimBlocksp>>>(r1_old->spec, r1_new->spec, r1_old->mesh->Nxh, r1_old->mesh->Ny, r1_old->mesh->BSZ);
        SpecSet<<<mesh->dimGridsp, mesh->dimBlocksp>>>(r2_old->spec, r2_new->spec, r2_old->mesh->Nxh, r2_old->mesh->Ny, r2_old->mesh->BSZ);
        // cout << "flag 12" << endl;
        // SpecSet<<<mesh->dimGridsp, mesh->dimBlocksp>>>(u->spec, unew->spec, mesh->Nxh, mesh->Ny, mesh->BSZ);
        // cuda_error_func( hipDeviceSynchronize() );

        if (m%10 == 0) {

        cout << "\r" << "t = " << m*dt << flush;
        }
        if (m%80 == 0){
            BwdTrans(mesh, r1_old->spec, r1_old->phys);
            BwdTrans(mesh, r2_old->spec, r2_old->phys);
            BwdTrans(mesh, w_old->spec, w_old->phys);
            cuda_error_func( hipDeviceSynchronize() );

            field_visual(r1_old, to_string(m+startpoint)+"r1.csv");
            field_visual(r2_old, to_string(m+startpoint)+"r2.csv");
            field_visual(w_old, to_string(m+startpoint)+"w.csv");
            if (std::isnan(r1_old->phys[0])) {"NAN ";exit(0);}
        }
        // if(m%100 == 0) cout << "t = " << m*dt << endl;
        // if (m%200 == 0){
        //     BwdTrans(mesh, u->spec, u->phys);
        //     cuda_error_func( hipDeviceSynchronize() );
        //     field_visual(u, to_string(m)+"u.csv");
        //     // printf("t: %f    val:%.8f   exa:%.8f    err: %.8f \n",m*dt,  u->phys[5],exact((m)*dt), u->phys[5]-exact((m)*dt));
        //     // cout<<"t: " << m*dt << "  " << u->phys[5] << endl;
        // }
    }

    return 0;
}

